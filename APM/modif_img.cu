#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include <cstdlib>
#include <cstdio>
#include "FreeImage.h"
#include <hip/hip_runtime.h>

#define WIDTH 1920
#define HEIGHT 1024
#define BPP 24 // Since we're outputting three 8 bit RGB values

using namespace std;

//Question 6
__global__ void saturate_component(unsigned int* c_d_img, int width, int height, int component) {


    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * 3;
        if (component == 0) {  // saturate red component
            c_d_img[idx] = 255;
        } else if (component == 1) {  // saturate green component
            c_d_img[idx + 1] = 255;
        } else {  // saturate blue component
            c_d_img[idx + 2] = 255;
        }
    }
}

//Question 7
__global__ void horizontal_flip(unsigned int* c_d_img, int width, int height)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < width / 2 && y < height)
    {
        int idx1 = (y * width + x) * 3;
        int idx2 = (y * width + (width - x - 1)) * 3;
        
        // Swap pixel values between idx1 and idx2
        unsigned int tmp;
        tmp = c_d_img[idx1]; c_d_img[idx1] = c_d_img[idx2]; c_d_img[idx2] = tmp;
        tmp = c_d_img[idx1+1]; c_d_img[idx1+1] = c_d_img[idx2+1]; c_d_img[idx2+1] = tmp;
        tmp = c_d_img[idx1+2]; c_d_img[idx1+2] = c_d_img[idx2+2]; c_d_img[idx2+2] = tmp;
    }

    __syncthreads();
}


int main (int argc , char** argv)
{
  FreeImage_Initialise();
  const char *PathName = "img.jpg";
  const char *PathDest = "new_img.png";
  // load and decode a regular file
  FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);

  FIBITMAP* bitmap = FreeImage_Load(FIF_JPEG, PathName, 0);

  if(! bitmap )
    exit( 1 ); //WTF?! We can't even allocate images ? Die !

  unsigned width  = FreeImage_GetWidth(bitmap);
  unsigned height = FreeImage_GetHeight(bitmap);
  unsigned pitch  = FreeImage_GetPitch(bitmap);

  fprintf(stderr, "Processing Image of size %d x %d\n", width, height);

  unsigned int *img = (unsigned int*) malloc(sizeof(unsigned int) * 3 * width * height);
  unsigned int *d_img = (unsigned int*) malloc(sizeof(unsigned int) * 3 * width * height);
  unsigned int *d_tmp = (unsigned int*) malloc(sizeof(unsigned int) * 3 * width * height);

  BYTE *bits = (BYTE*)FreeImage_GetBits(bitmap);
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = (BYTE*)bits;
    for ( int x =0; x<width; x++)
    {
      int idx = ((y * width) + x) * 3;
      img[idx + 0] = pixel[FI_RGBA_RED];
      img[idx + 1] = pixel[FI_RGBA_GREEN];
      img[idx + 2] = pixel[FI_RGBA_BLUE];
      pixel += 3;
    }
    // next line
    bits += pitch;
  }

  memcpy(d_img, img, 3 * width * height * sizeof(unsigned int));
  memcpy(d_tmp, img, 3 * width * height * sizeof(unsigned int));

  unsigned int  *c_d_img;

  hipMalloc((void **)&c_d_img, sizeof(unsigned int) * width * height * 3);
  hipMemcpy(c_d_img, img, sizeof(unsigned int) * width * height * 3, hipMemcpyHostToDevice);


  // Kernel
  dim3 block_size(32, 32);
  dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);

  saturate_component<<<grid_size, block_size>>>(c_d_img, width, height, 0);
  //horizontal_flip<<<grid_size, block_size>>>(c_d_img, width, height);


  hipMemcpy(d_img, c_d_img, sizeof(unsigned int) * 3 * width * height, hipMemcpyDeviceToHost);
  
  // Copy back
  memcpy(img, d_img, 3 * width * height * sizeof(unsigned int));

  bits = (BYTE*)FreeImage_GetBits(bitmap);
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = (BYTE*)bits;
    for ( int x =0; x<width; x++)
    {
      RGBQUAD newcolor;

      int idx = ((y * width) + x) * 3;
      newcolor.rgbRed = img[idx + 0];
      newcolor.rgbGreen = img[idx + 1];
      newcolor.rgbBlue = img[idx + 2];

      if(!FreeImage_SetPixelColor(bitmap, x, y, &newcolor))
      { fprintf(stderr, "(%d, %d) Fail...\n", x, y); }

      pixel+=3;
    }
    // next line
    bits += pitch;
  }

  if( FreeImage_Save (FIF_PNG, bitmap , PathDest , 0 ))
    cout << "Image successfully saved ! " << endl ;
  FreeImage_DeInitialise(); //Cleanup !

  free(img);
  free(d_img);
  free(d_tmp);
  hipFree(c_d_img);
}
